#include "hip/hip_runtime.h"
// Copyright(c) 2022 Alex S.Fu All rights reserved.
// Inspired by https://github.com/IwakuraRein/KernelFilter-PyTorch/tree/main/_KernelFilter
// Copyright (c) 1993-2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.

#include "kpn_plugin.h"
#include "tensorRT/plugin/common/trt_common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace nvinfer1 {


template<typename scalar_t>
__global__ void kernelFilterKernel(
    const scalar_t *__restrict__ kernel,  // [B, H, W, k, k]
    const scalar_t *__restrict__ radiance,// [B, H, W, C]
    scalar_t *__restrict__ output,        // [B, H, W, C]
    const int batch,
    const int channel,
    const int height,
    const int width,
    const int k0,    // filter size
    const int half_k,// k0 / 2
    const int dilation_h,
    const int dilation_w) {

    // batch index
    const int n = blockIdx.y;
    // column index
    const int col = blockIdx.x * blockDim.x + threadIdx.x;

    const int xi = col / width;// Kernel height index
    const int yi = col % width;// Kernel width index

    // radiance offset
    const int off_r = n * height * width * channel;

    // kernel radiance offset
    const int off_k = ((n * height + xi) * width + yi) * k0 * k0;

    if (n >= batch || xi >= height || yi >= width) return;
    scalar_t color[3] = { scalar_t(0), scalar_t(0), scalar_t(0) };
    
    for (int io = -half_k; io <= half_k; io++) {
        int xo = xi + io * dilation_h;// True output height index
        if (xo < 0 || xo >= height) continue;
        for (int jo = -half_k; jo <= half_k; jo++) {
            int yo = yi + jo * dilation_w;// True output width index
            if (yo < 0 || yo >= width) continue;
            int radiance_idx = off_r + (xo * width + yo) * channel;
            int kernel_idx = off_k + (io + half_k) * k0 + jo + half_k;
            #pragma unroll
            for (int c = 0; c < 3; c++) {
                if (radiance[radiance_idx + c] > scalar_t(0)) {
                    color[c] += radiance[radiance_idx + c] * kernel[kernel_idx];
                }
            }
        }
    }
    int output_idx = off_r + (xi * width + yi) * channel;
    output[output_idx] = color[0];
    output[output_idx + 1] = color[1];
    output[output_idx + 2] = color[2];
}

/**
  *
  */
template<typename scalar_t>
int kernelFilterKernelLauncher(
    const scalar_t *__restrict__ kernel,  // [B, H, W, k, k]
    const scalar_t *__restrict__ radiance,// [B, C, H, W]
    scalar_t *__restrict__ output,        // [B, C, H, W]
    const int batch,
    const int channel,
    const int height,
    const int width,
    const int k0,    // filter size
    const int half_k,// k0 / 2
    const int dilation_h,
    const int dilation_w) {
    const dim3 blocks(GET_BLOCKS(height * width), batch);

    kernelFilterKernel<scalar_t><<<blocks, NUM_THREADS>>>(
        kernel,
        radiance,
        output,
        batch,
        channel,
        height,
        width,
        k0,
        half_k,
        dilation_h,
        dilation_w);
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n", __FILE__, __LINE__, hipGetErrorString(err));
        return 1;
    }
    return 0;
}

// TODO: Int8
int32_t KPNPluginDynamic::enqueue(const nvinfer1::PluginTensorDesc *inputDesc,
                              const nvinfer1::PluginTensorDesc *outputDesc,
                              const void *const *inputs, void *const *outputs,
                              void *workSpace, hipStream_t stream) TRT_NOEXCEPT {

    int k = inputDesc[0].dims.d[3];

    int rb = inputDesc[1].dims.d[0];
    int rc = inputDesc[1].dims.d[1];
    int rh = inputDesc[1].dims.d[2];
    int rw = inputDesc[1].dims.d[3];

    const void *kernel = inputs[0];
    const void *radiance = inputs[1];

    void *output = outputs[0];

    // quantization
    float inputScale = inputDesc->scale;
    float outputScale = outputDesc->scale;

    auto data_type = inputDesc[0].type;
    switch (data_type) {
        case nvinfer1::DataType::kFLOAT:
            return kernelFilterKernelLauncher<float>(
                (float *)kernel,
                (float *)radiance,
                (float *)output,
                rb, rc, rh, rw, k, k / 2, mDilation.d[0], mDilation.d[1]);
            break;
        case nvinfer1::DataType::kHALF:
            return kernelFilterKernelLauncher<__half>(
                (__half *)kernel,
                (__half *)radiance,
                (__half *)output,
                rb, rc, rh, rw, k, k / 2, mDilation.d[0], mDilation.d[1]);
        case nvinfer1::DataType::kINT8:// TODO quantization
            return 1;
        default:
            return 1;
    }
}

}

